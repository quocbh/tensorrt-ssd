#include "hip/hip_runtime.h"
/*
 * http://github.com/dusty-nv/jetson-inference
 */

#include "mathFunctions.h"
#include <iostream>
#include "../util/cuda/cudaUtility.h"



template <typename Dtype>
__global__ void Concat(const int nthreads, const Dtype* in_data,
                       const bool forward, const int num_concats, const int concat_size,
                       const int top_concat_axis, const int bottom_concat_axis,
                       const int offset_concat_axis, Dtype* out_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int total_concat_size = concat_size * bottom_concat_axis;
        const int concat_num = index / total_concat_size;
        const int concat_index = index % total_concat_size;
        const int top_index = concat_index +
                              (concat_num * top_concat_axis + offset_concat_axis) * concat_size;
        if (forward) {
            out_data[top_index] = in_data[index];
        } else {
            out_data[index] = in_data[top_index];
        }
    }
}
hipError_t ConcatLayer(int nthreads, const float *bottom_data, bool kForward, int num_concats_, int concat_input_size_,
                        int top_concat_axis, int bottom_concat_axis, int offset_concat_axis, float *top_data, hipStream_t stream)
{
    Concat<float><<<TENSORRT_GET_BLOCKS(nthreads), TENSORRT_CUDA_NUM_THREADS,0,stream>>>(nthreads, bottom_data,
    kForward, num_concats_, concat_input_size_, top_concat_axis, bottom_concat_axis, offset_concat_axis, top_data);
    return hipPeekAtLastError();
}


// gpuPreImageNet
__global__ void gpuPreImageNet( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = oWidth * oHeight;
	
	if( x >= oWidth || y >= oHeight )
		return;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.z, px.y, px.x);
	
	output[n * 0 + y * oWidth + x] = bgr.x;
	output[n * 1 + y * oWidth + x] = bgr.y;
	output[n * 2 + y * oWidth + x] = bgr.z;
}

// cudaPreImageNet
hipError_t cudaPreImageNet( float4* input, size_t inputWidth, size_t inputHeight,
				         float* output, size_t outputWidth, size_t outputHeight )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNet<<<gridDim, blockDim>>>(scale, input, inputWidth, output, outputWidth, outputHeight);

	return CUDA(hipGetLastError());
}

// gpuPreImageNetMean
__global__ void gpuPreImageNetMean( float2 scale, float3* input, int iWidth, float* output, int oWidth, int oHeight, float3 mean_value )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = oWidth * oHeight;
	
	if( x >= oWidth || y >= oHeight )
		return;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float3 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.z - mean_value.x, px.y - mean_value.y, px.x - mean_value.z);
	
	output[n * 0 + y * oWidth + x] = bgr.x;
	output[n * 1 + y * oWidth + x] = bgr.y;
	output[n * 2 + y * oWidth + x] = bgr.z;
}

// cudaPreImageNetMean
hipError_t cudaPreImageNetMean( float3* input, size_t inputWidth, size_t inputHeight,
				             float* output, size_t outputWidth, size_t outputHeight, const float3& mean_value )

{
	if( !input || !output ){
        std::cout << "error here. "<< std::endl;
        return hipErrorInvalidDevicePointer;
    }

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 ){
        std::cout << "Or here. " << std::endl;
        return hipErrorInvalidValue;
    }


	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );


	// launch kernel

	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetMean<<<gridDim, blockDim>>>(scale, input, inputWidth, output, outputWidth, outputHeight, mean_value);

	return CUDA(hipGetLastError());

}

__global__ void kernel_extract_roi(float* input, float* output, char* mean,
    const int input_w, const int output_w, const int output_h,
    const int in_plane_r, const int in_plane_g, const int in_plane_b,
    const int out_plane_r, const int out_plane_g, const int out_plane_b,
    const int bbox_x, const int bbox_y, const int bbox_w, const int bbox_h)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if( x < output_w && y < output_h)
    {
        float r[2] = { float(x) * bbox_w / output_w + bbox_x,
                       float(y) * bbox_h / output_h + bbox_y };

        int   pos[4][2] = { { int(floor(r[0])), int(floor(r[1])) },
                            { int( ceil(r[0])), int(floor(r[1])) },
                            { int(floor(r[0])),  int(ceil(r[1])) },
                            { int( ceil(r[0])),  int(ceil(r[1])) } };

        float u = r[0]-floor(r[0]);
        float v = r[1]-floor(r[1]);

        float s[4] = { (1-u)*(1-v), u*(1-v), (1-u)*v, u*v };

        int map[4] = { pos[0][1]*input_w + pos[0][0], pos[1][1]*input_w + pos[1][0],
                       pos[2][1]*input_w + pos[2][0], pos[3][1]*input_w + pos[3][0]};

        int idx = y * output_w + x;
        output[idx+out_plane_r] = round( s[0]*input[map[0]+in_plane_r]
                                       + s[1]*input[map[1]+in_plane_r]
                                       + s[2]*input[map[2]+in_plane_r]
                                       + s[3]*input[map[3]+in_plane_r] );// float(mean[idx+out_plane_r]));
        output[idx+out_plane_g] = round( s[0]*input[map[0]+in_plane_g]
                                       + s[1]*input[map[1]+in_plane_g]
                                       + s[2]*input[map[2]+in_plane_g]
                                       + s[3]*input[map[3]+in_plane_g] );//float(mean[idx+out_plane_g]));
        output[idx+out_plane_b] = round( s[0]*input[map[0]+in_plane_b]
                                       + s[1]*input[map[1]+in_plane_b]
                                       + s[2]*input[map[2]+in_plane_b]
                                       + s[3]*input[map[3]+in_plane_b] );//float(mean[idx+out_plane_b]));
    }
}


__global__  void kernelSoftmax( float* x, int channels, float* y)
{

    extern __shared__ float mem[];
    __shared__ float sum_value;

    float number = *(x + blockDim.x*blockIdx.x + threadIdx.x);
    float number_exp = __expf(number);


    atomicAdd(&sum_value, number_exp);
    __syncthreads();


    y[blockDim.x*blockIdx.x + threadIdx.x] = __fdiv_rd(number_exp, sum_value);

}

void cudaSoftmax(int n, int channels,  float* x, float*y)
{
	kernelSoftmax<<< (n/channels), channels, channels*sizeof(float)>>>( x, channels, y);
	hipDeviceSynchronize();
}

